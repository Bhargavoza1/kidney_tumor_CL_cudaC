 
#include "Tensor.h"
#include <vector>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cassert>
namespace Hex {
    // Constructor
    template <typename T>
    Tensor<T>::Tensor(const std::vector<int>& shape, bool iscudafree) : shape(shape) ,_iscudafree(iscudafree) {
       
        int size = 1;
        for (int dim : shape) {
            size *= dim;
        }
        T* cudaData;
        hipMalloc((void**)&cudaData, size * sizeof(T));
        data = std::shared_ptr<T[]>(cudaData, [=](T* ptr) { if (iscudafree) { hipFree(ptr); } }); // Custom deleter for CUDA memory
    }

    // Destructor
    template <typename T>
    Tensor<T>::~Tensor() {
       //  cudafree();
    }
    template <typename T>
    void Tensor<T>::cudafree() { if (this != nullptr) { if (_iscudafree) { hipFree(data.get()); }}}
 
 
    // Set element at index
    template <typename T>
    void Tensor<T>::set(const std::vector<int>& indices, T value) {
        int index = calculateIndex(indices);
        hipMemcpy(data.get() + index, &value, sizeof(T), hipMemcpyHostToDevice);
    }

    // Get element at index
    template <typename T>
    T Tensor<T>::get(const std::vector<int>& indices) const {
        int index = calculateIndex(indices);
        T value;
        hipMemcpy(&value, data.get() + index, sizeof(T), hipMemcpyDeviceToHost);
        return value;
    }

    // Print the tensor
    template <typename T>
    void Tensor<T>::print() const {
        std::cout << "Tensor (Shape: ";
        for (size_t i = 0; i < shape.size(); ++i) {
            std::cout << shape[i];
            if (i < shape.size() - 1) {
                std::cout << "x";
            }
        }
        std::cout << ", Type: " << typeid(T).name() << "):" << std::endl;

        printHelper(data.get(), shape, 0, {});
        //std::cout << std::endl;
    }

    template <typename T>
    void Tensor<T>::printshape() const {
        std::cout << "Tensor (Shape: ";
        for (size_t i = 0; i < shape.size(); ++i) {
            std::cout << shape[i];
            if (i < shape.size() - 1) {
                std::cout << "x";
            }
        }
        std::cout << ", Type: " << typeid(T).name() << "):" << std::endl;
    }

    template <typename T>
    void Tensor<T>::setData(T* newData) {
        int size = 1;
        for (int dim : shape) {
            size *= dim;
        }
        //  T* cudaData;
         // hipMalloc((void**)&cudaData, size * sizeof(T));
         // data = std::shared_ptr<T[]>(cudaData, [=](T* ptr) { if (_iscudafree) { hipFree(ptr); } }); // Custom deleter for CUDA memory
        hipMemcpy(data.get(), newData, size * sizeof(T), hipMemcpyDeviceToDevice);
    }

    // Getter for shape
    template <typename T>
    std::vector<int> Tensor<T>::getShape() const {
        return shape;
    }

    template <typename T>
    const T* Tensor<T>::getData() const {
        return data.get();
    }

    template <typename T>
    T* Tensor<T>::getData() {
        return data.get();
    }


    // Helper function to calculate the flat index from indices
    template <typename T>
    int Tensor<T>::calculateIndex(const std::vector<int>& indices) const {
        int index = 0;
        int stride = 1;
        for (int i = shape.size() - 1; i >= 0; --i) {
            index += indices[i] * stride;
            stride *= shape[i];
        }
        return index;
    }

    // Helper function to print tensor data recursively
    template <typename T>
    void Tensor<T>::printHelper(const T* data, const std::vector<int>& shape, int dimension, std::vector<int> indices) const {
        int currentDimensionSize = shape[dimension];

        std::cout << "[";

        for (int i = 0; i < currentDimensionSize; ++i) {
            indices.push_back(i);

            if (dimension < shape.size() - 1) {
                // If not the last dimension, recursively print the next dimension
                printHelper(data, shape, dimension + 1, indices);
            }
            else {
                // If the last dimension, print the actual element
                std::cout << get(indices);
            }

            indices.pop_back();

            if (i < currentDimensionSize - 1) {
                std::cout << ", ";
            }
        }

        std::cout << "]";

        if (dimension < shape.size() - 1) {
            // If not the last dimension, add a new line after completing the inner block
            std::cout << std::endl;
        }
    }


    template <typename T>
    void Tensor<T>::reshape(const std::vector<int>& new_shape) {
        int new_size = 1;
        for (int dim : new_shape) {
            new_size *= dim;   
        }
 
        int size = 1;
        for (int dim : getShape() ) {
            size *= dim; 
        }


        if (new_size != size ) { 
            assert(false && "Error: New shape's total size does not match current size.");
            return;
        }
        shape = new_shape;
    }
     

    // Helper function to calculate indices from a flat index
     template <typename T>
     std::vector<int> Tensor<T>::calculateIndices(int index) const {
        std::vector<int> indices(shape.size(), 0);
        for (int i = shape.size() - 1; i >= 0; --i) {
            indices[i] = index % shape[i];
            index /= shape[i];
        }
        return indices;
     }
    // Other member function definitions...

    // Explicit instantiation of the template class for supported types
    template class Tensor<float>;
    template class Tensor<int>;
    template class Tensor<double>;
}
